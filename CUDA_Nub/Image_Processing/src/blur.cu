#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

#include "../include/utils.h"
#include "../include/loadSaveImage.h"

static const int filterWidth = 9;
static const float filterSigma = 2.f;

void preProcess(uchar4 **h_inputImageRGBA, uchar4 **h_outputImageRGBA,
                uchar4 **d_inputImageRGBA, uchar4 **d_outputImageRGBA,
                unsigned char **d_redBlurred, unsigned char **d_red,
                unsigned char **d_greenBlurred, unsigned char **d_green,
                unsigned char **d_blueBlurred, unsigned char **d_blue,
                float **h_filter, float **d_filter,
                size_t &rows, size_t &cols,
                const std::string &filename){
    checkCudaErrors(hipFree(0));
	
    loadImageRGBA(filename, h_inputImageRGBA, &rows, &cols);
	
    *h_outputImageRGBA = new uchar4[rows * cols];
	
    size_t numPixels = rows * cols;
    checkCudaErrors(hipMalloc(d_inputImageRGBA, sizeof(uchar4) * numPixels));
    checkCudaErrors(hipMalloc(d_outputImageRGBA, sizeof(uchar4) * numPixels));

    checkCudaErrors(hipMemcpy(*d_inputImageRGBA, *h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(*d_outputImageRGBA, 0, numPixels * sizeof(uchar4)));
	
    *h_filter = new float[filterWidth * filterWidth];

    float filterSum = 0.f; 
    for (int r = -filterWidth / 2; r <= filterWidth / 2; ++r){
        for (int c = -filterWidth / 2; c <= filterWidth / 2; ++c){
            float filterValue = expf(-(float)(c * c + r * r) / (2.f * filterSigma * filterSigma));
            (*h_filter)[(r + filterWidth / 2) * filterWidth + c + filterWidth / 2] = filterValue;
            filterSum += filterValue; // for normalization
        }
    }
	
    float normalizationFactor = 1.f / filterSum;
    for (int r = -filterWidth / 2; r <= filterWidth / 2; ++r)
        for (int c = -filterWidth / 2; c <= filterWidth / 2; ++c)
            (*h_filter)[(r + filterWidth / 2) * filterWidth + c + filterWidth / 2] *= normalizationFactor;
			
    checkCudaErrors(hipMalloc(d_red, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMalloc(d_green, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMalloc(d_blue, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMalloc(d_redBlurred, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMalloc(d_greenBlurred, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMalloc(d_blueBlurred, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMemset(*d_redBlurred, 0, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMemset(*d_greenBlurred, 0, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMemset(*d_blueBlurred, 0, sizeof(unsigned char) * numPixels));
    checkCudaErrors(hipMalloc(d_filter, sizeof(float) * filterWidth * filterWidth));
    checkCudaErrors(hipMemcpy(*d_filter, *h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void postProcess(const std::string &output_file, uchar4 *const h_outputImage, const uchar4 *const d_outputImage,
                 const int rows, const int cols){
    size_t numPixels = rows * cols;
    checkCudaErrors(hipMemcpy(h_outputImage, d_outputImage, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost));
    saveImageRGBA(h_outputImage, rows, cols, output_file);
}

__global__ void gaussian_blur_kernel(const unsigned char *const inputChannel,
                                     unsigned char *const outputChannel,
                                     const int numRows, const int numCols,
                                     const float *const filter, const int filterWidth){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= numCols || i >= numRows)
        return;

    float result = 0.f;
    for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; ++filter_r){
        for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; ++filter_c){
            int image_r = min(max(i + filter_r, 0), numRows - 1);
            int image_c = min(max(j + filter_c, 0), numCols - 1);

            float image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
            float filter_value = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];

            result += image_value * filter_value;
        }
    }
    outputChannel[i * numCols + j] = static_cast<unsigned char>(result);
}

__global__ void separateChannels_kernel(const uchar4 *const inputImageRGBA,
                                        const int numRows, const int numCols,
                                        unsigned char *const redChannel,
                                        unsigned char *const greenChannel,
                                        unsigned char *const blueChannel){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= numCols || i >= numRows)
        return;

    int tid = i * numCols + j;

    redChannel[tid] = inputImageRGBA[tid].x;
    greenChannel[tid] = inputImageRGBA[tid].y;
    blueChannel[tid] = inputImageRGBA[tid].z;
}

__global__ void recombineChannels_kernel(const unsigned char *const redChannel,
                                         const unsigned char *const greenChannel,
                                         const unsigned char *const blueChannel,
                                         uchar4 *const outputImageRGBA,
                                         const int numRows, const int numCols){
    int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                   blockIdx.y * blockDim.y + threadIdx.y);

    int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        return;

    unsigned char red = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue = blueChannel[thread_1D_pos];
	
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);

    outputImageRGBA[thread_1D_pos] = outputPixel;
}

void cuda_gaussian_blur(const uchar4 *const h_inputImageRGBA, uchar4 *const d_inputImageRGBA,
                        uchar4 *const d_outputImageRGBA, const int numRows, const int numCols,
                        unsigned char *d_redBlurred, unsigned char *d_red,
                        unsigned char *d_greenBlurred, unsigned char *d_green,
                        unsigned char *d_blueBlurred, unsigned char *d_blue,
                        float *d_filter, const int filterWidth){
    int blockW = 32;
    int blockH = 32;

    dim3 blockSize(blockW, blockH);
	
    int gridW = (numCols % blockW != 0) ? (numCols / blockW + 1) : (numCols / blockW);
    int gridH = (numRows % blockH != 0) ? (numRows / blockH + 1) : (numRows / blockH);
    dim3 gridSize(gridW, gridH);

    separateChannels_kernel<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
	
    gaussian_blur_kernel<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur_kernel<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur_kernel<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    recombineChannels_kernel<<<gridSize, blockSize>>>(d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputImageRGBA, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}

void gaussian_blur(const std::string &input_file, const std::string &output_file){
    size_t numRows, numCols;

    uchar4 *h_inputImageRGBA, *d_inputImageRGBA;
    uchar4 *h_outputImageRGBA, *d_outputImageRGBA;
    unsigned char *d_redBlurred, *d_greenBlurred, *d_blueBlurred;
    unsigned char *d_red, *d_green, *d_blue;
    float *h_filter, *d_filter;

    preProcess(&h_inputImageRGBA, &h_outputImageRGBA, &d_inputImageRGBA, &d_outputImageRGBA,
               &d_redBlurred, &d_red, &d_greenBlurred, &d_green, &d_blueBlurred, &d_blue,
               &h_filter, &d_filter, numRows, numCols, input_file);

    cuda_gaussian_blur(h_inputImageRGBA, d_inputImageRGBA, d_outputImageRGBA, numRows, numCols,
                       d_redBlurred, d_greenBlurred, d_blueBlurred, d_red, d_green, d_blue,
                       d_filter, filterWidth);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    postProcess(output_file, h_outputImageRGBA, d_outputImageRGBA, numRows, numCols);

    checkCudaErrors(hipFree(d_inputImageRGBA));
    checkCudaErrors(hipFree(d_outputImageRGBA));
    checkCudaErrors(hipFree(d_filter));
    checkCudaErrors(hipFree(d_red));
    checkCudaErrors(hipFree(d_green));
    checkCudaErrors(hipFree(d_blue));
    checkCudaErrors(hipFree(d_redBlurred));
    checkCudaErrors(hipFree(d_greenBlurred));
    checkCudaErrors(hipFree(d_blueBlurred));
    delete[] h_inputImageRGBA;
    delete[] h_outputImageRGBA;
    delete[] h_filter;
}